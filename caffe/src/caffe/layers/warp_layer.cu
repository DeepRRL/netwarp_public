#include "hip/hip_runtime.h"
// Copyright 2017 Max Planck Society
// Distributed under the BSD-3 Software license,
// (See accompanying file LICENSE.txt or copy at
// https://opensource.org/licenses/BSD-3-Clause)
#include <cfloat>
#include <vector>

#include "caffe/layers/warp_layer.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/util/gpu_util.cuh"

namespace caffe {

template <typename Dtype>
__global__ void truncate_interp2_fwd(const int nthreads, const Dtype *bottom_0_data_, const Dtype *bottom_1_data_,
                                     const int num_, const int channels_, const int height_, const int width_, 
                                     Dtype *theta_data, Dtype* theta_data_, Dtype *x_w_data, Dtype *top_data) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    int temp = 0;
    const int n = index / (channels_ * height_ * width_);
    temp = index % (channels_ * height_ * width_);   
    const int c = temp / (height_ * width_);
    temp = temp % (height_ * width_);
    const int h = temp / width_;
    const int w = temp % width_;
    int index_x = ((n * 2 + 1) * height_ + h) * width_ + w;
    int index_y = ((n * 2 + 0) * height_ + h) * width_ + w;
    x_w_data[ index_x ] = h + bottom_1_data_[ index_x ];
    x_w_data[ index_y ] = w + bottom_1_data_[ index_y ];
    int xw_floor = (int)floor(x_w_data[ index_x ]);
    int yw_floor = (int)floor(x_w_data[ index_y ]);
    int xw_ceil = (int)ceil(x_w_data[ index_x ]);
    int yw_ceil = (int)ceil(x_w_data[ index_y ]);
    theta_data[ index_x ] = x_w_data[ index_x ] - floor(x_w_data[ index_x ]);
    theta_data[ index_y ] = x_w_data[ index_y ] - floor(x_w_data[ index_y ]);
    theta_data_[ index_x ] = 1 - theta_data[ index_x ];
    theta_data_[ index_y ] = 1 - theta_data[ index_y ];
    int offset = (n * channels_ + c) * height_;
    if (x_w_data[ index_x ] >= 0 && x_w_data[ index_x ] <= height_-1 && 
        x_w_data[ index_y ] >= 0 && x_w_data[ index_y ] <= width_-1) {
      Dtype I0 = bottom_0_data_[ (offset + xw_floor) * width_ + yw_floor ]; 
      Dtype I1 = bottom_0_data_[ (offset + xw_ceil ) * width_ + yw_floor ]; 
      Dtype I2 = bottom_0_data_[ (offset + xw_floor) * width_ + yw_ceil ]; 
      Dtype I3 = bottom_0_data_[ (offset + xw_ceil ) * width_ + yw_ceil ];
      top_data[ (offset +  h) * width_ +  w ] = (theta_data_[index_x] * theta_data_[index_y] * I0) + 
                                                (theta_data[index_x]  * theta_data_[index_y] * I1) + 
                                                (theta_data_[index_x] * theta_data[index_y]  * I2) + 
                                                (theta_data[index_x]  * theta_data[index_y]  * I3);
    }
  }
}

template <typename Dtype>
__global__ void nearest_interp2_fwd(const int nthreads, const Dtype *bottom_0_data_, const Dtype *bottom_1_data_,
                                    const int num_, const int channels_, const int height_, const int width_, 
                                    Dtype *theta_data, Dtype* theta_data_, Dtype *x_w_data, Dtype *top_data) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    int temp = 0;
    const int n = index / (channels_ * height_ * width_);
    temp = index % (channels_ * height_ * width_);   
    const int c = temp / (height_ * width_);
    temp = temp % (height_ * width_);
    const int h = temp / width_;
    const int w = temp % width_;
    int index_x = ((n * 2 + 1) * height_ + h) * width_ + w;
    int index_y = ((n * 2 + 0) * height_ + h) * width_ + w;
    x_w_data[ index_x ] = h + bottom_1_data_[ index_x ];
    x_w_data[ index_y ] = w + bottom_1_data_[ index_y ];
    int xw_floor = (int)floor(x_w_data[ index_x ]);
    int yw_floor = (int)floor(x_w_data[ index_y ]);
    int xw_ceil = (int)ceil(x_w_data[ index_x ]);
    int yw_ceil = (int)ceil(x_w_data[ index_y ]);
    theta_data[ index_x ] = x_w_data[ index_x ] - floor(x_w_data[ index_x ]);
    theta_data[ index_y ] = x_w_data[ index_y ] - floor(x_w_data[ index_y ]);
    if (x_w_data[ index_x ] < 0) {
      theta_data[ index_x ] = x_w_data[ index_x ];
      xw_floor = 0; xw_ceil = 0;
    } 
    if (x_w_data[ index_x ] >= height_-1) {
      theta_data[ index_x ] = x_w_data[ index_x ] - height_;
      xw_floor = height_-1; xw_ceil = height_-1;
    }
    if (x_w_data[ index_y ] < 0) {
      theta_data[ index_y ] = x_w_data[ index_y ];
      yw_floor = 0; yw_ceil = 0;
    }
    if (x_w_data[ index_y ] >= width_-1) {
      theta_data[ index_y ] = x_w_data[ index_y ] - width_;
      yw_floor = width_-1; yw_ceil = width_-1;
    }
    theta_data_[ index_x ] = 1 - theta_data[ index_x ];
    theta_data_[ index_y ] = 1 - theta_data[ index_y ];
    int offset = (n * channels_ + c) * height_;
    Dtype I0 = bottom_0_data_[ (offset + xw_floor) * width_ + yw_floor ]; 
    Dtype I1 = bottom_0_data_[ (offset + xw_ceil ) * width_ + yw_floor ]; 
    Dtype I2 = bottom_0_data_[ (offset + xw_floor) * width_ + yw_ceil ]; 
    Dtype I3 = bottom_0_data_[ (offset + xw_ceil ) * width_ + yw_ceil ];
    top_data[ (offset +  h) * width_ +  w ] = (theta_data_[index_x] * theta_data_[index_y] * I0) + 
                                              (theta_data[index_x]  * theta_data_[index_y] * I1) + 
                                              (theta_data_[index_x] * theta_data[index_y]  * I2) + 
                                              (theta_data[index_x]  * theta_data[index_y]  * I3);
  }
}

template <typename Dtype>
__global__ void truncate_interp2_bwd(const int nthreads, const int num_, const int channels_, const int height_, 
                                     const int width_, const Dtype *theta_data, const Dtype* theta_data_, 
                                     const Dtype *x_w_data, Dtype *bottom_0_diff, Dtype *bottom_1_diff, 
                                     const Dtype *top_diff, const Dtype *top_data, const Dtype *bottom_0_data) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    int temp = 0;
    const int n = index / (channels_ * height_ * width_);
    temp = index % (channels_ * height_ * width_);   
    const int c = temp / (height_ * width_);
    temp = temp % (height_ * width_);
    const int h = temp / width_;
    const int w = temp % width_;

    int index_x = ((n * 2 + 1) * height_ + h) * width_ + w;
    int index_y = ((n * 2 + 0) * height_ + h) * width_ + w;

    if (!(x_w_data[ index_x ] < 0 || x_w_data[ index_x ] > height_-1 ||
          x_w_data[ index_y ] < 0 || x_w_data[ index_y ] > width_-1)) {
    
      int xw_floor = (int)floor(x_w_data[ index_x ]);
      int yw_floor = (int)floor(x_w_data[ index_y ]);
      int xw_ceil = (int)ceil(x_w_data[ index_x ]);
      int yw_ceil = (int)ceil(x_w_data[ index_y ]);
  
      int bottom_0_index = ((n * channels_ + c) * height_ +  h) * width_ +  w;
      int offset = (n * channels_ + c) * height_;
      Dtype I0 = bottom_0_data[ (offset + xw_floor) * width_ + yw_floor ]; 
      Dtype I1 = bottom_0_data[ (offset + xw_ceil ) * width_ + yw_floor ]; 
      Dtype I2 = bottom_0_data[ (offset + xw_floor) * width_ + yw_ceil ]; 
      Dtype I3 = bottom_0_data[ (offset + xw_ceil ) * width_ + yw_ceil ];
      bottom_1_diff[ index_x ] += ( -1*theta_data_[index_y]*I0 + 
                                       theta_data_[index_y]*I1 - 
                                       theta_data[index_y] *I2 + 
                                       theta_data[index_y] *I3 ) * 
                                  top_diff[(offset + h) * width_ + w];
      bottom_1_diff[ index_y ] += ( -1*theta_data_[index_x]*I0 - 
                                       theta_data[index_x] *I1 + 
                                       theta_data_[index_x]*I2 + 
                                       theta_data[index_x] *I3 ) * 
                                  top_diff[(offset + h) * width_ + w];

      caffe_gpu_atomic_add((Dtype) theta_data_[ index_x ]*theta_data_[ index_y ]*top_diff[bottom_0_index], 
                           bottom_0_diff + ((offset + xw_floor) * width_ + yw_floor ));
      caffe_gpu_atomic_add((Dtype) theta_data[ index_x ] *theta_data_[ index_y ]*top_diff[bottom_0_index],
                           bottom_0_diff + ((offset + xw_ceil ) * width_ + yw_floor ));
      caffe_gpu_atomic_add((Dtype) theta_data_[ index_x ]*theta_data[ index_y ] *top_diff[bottom_0_index],
                           bottom_0_diff + ((offset + xw_floor) * width_ + yw_ceil  ));
      caffe_gpu_atomic_add((Dtype) theta_data[ index_x ] *theta_data[ index_y ] *top_diff[bottom_0_index], 
                           bottom_0_diff + ((offset + xw_ceil ) * width_ + yw_ceil  ));
    }
  }
}
 
template <typename Dtype>
__global__ void nearest_interp2_bwd(const int nthreads, const int num_, const int channels_, const int height_, 
                                    const int width_, const Dtype *theta_data, const Dtype* theta_data_, 
                                    const Dtype *x_w_data, Dtype *bottom_0_diff, Dtype *bottom_1_diff, 
                                    const Dtype *top_diff, const Dtype *top_data, const Dtype *bottom_0_data) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    int temp = 0;
    const int n = index / (channels_ * height_ * width_);
    temp = index % (channels_ * height_ * width_);   
    const int c = temp / (height_ * width_);
    temp = temp % (height_ * width_);
    const int h = temp / width_;
    const int w = temp % width_;

    int index_x = ((n * 2 + 1) * height_ + h) * width_ + w;
    int index_y = ((n * 2 + 0) * height_ + h) * width_ + w;

    int xw_floor = (int)floor(x_w_data[ index_x ]);
    int yw_floor = (int)floor(x_w_data[ index_y ]);
    int xw_ceil = (int)ceil(x_w_data[ index_x ]);
    int yw_ceil = (int)ceil(x_w_data[ index_y ]);

    if (x_w_data[ index_x ] < 0) {
      xw_floor = 0; xw_ceil = 0;
    } 
    if (x_w_data[ index_x ] >= height_-1) {
      xw_floor = height_-1; xw_ceil = height_-1;
    }
    if (x_w_data[ index_y ] < 0) {
      yw_floor = 0; yw_ceil = 0;
    }
    if (x_w_data[ index_y ] >= width_-1) {
      yw_floor = width_-1; yw_ceil = width_-1;
    }

    int bottom_0_index = ((n * channels_ + c) * height_ +  h) * width_ +  w;
    int offset = (n * channels_ + c) * height_;
    Dtype I0 = bottom_0_data[ (offset + xw_floor) * width_ + yw_floor ]; 
    Dtype I1 = bottom_0_data[ (offset + xw_ceil ) * width_ + yw_floor ]; 
    Dtype I2 = bottom_0_data[ (offset + xw_floor) * width_ + yw_ceil ]; 
    Dtype I3 = bottom_0_data[ (offset + xw_ceil ) * width_ + yw_ceil ];
    bottom_1_diff[ index_x ] += ( -1*theta_data_[index_y]*I0 + 
                                     theta_data_[index_y]*I1 - 
                                     theta_data[index_y] *I2 + 
                                     theta_data[index_y] *I3 ) * 
                                top_diff[(offset + h) * width_ + w];
    bottom_1_diff[ index_y ] += ( -1*theta_data_[index_x]*I0 - 
                                     theta_data[index_x] *I1 + 
                                     theta_data_[index_x]*I2 + 
                                     theta_data[index_x] *I3 ) * 
                                top_diff[(offset + h) * width_ + w];
    caffe_gpu_atomic_add((Dtype) theta_data_[ index_x ]*theta_data_[ index_y ]*top_diff[bottom_0_index], 
                         bottom_0_diff + ((offset + xw_floor) * width_ + yw_floor ));
    caffe_gpu_atomic_add((Dtype) theta_data[ index_x ] *theta_data_[ index_y ]*top_diff[bottom_0_index],
                         bottom_0_diff + ((offset + xw_ceil ) * width_ + yw_floor ));
    caffe_gpu_atomic_add((Dtype) theta_data_[ index_x ]*theta_data[ index_y ] *top_diff[bottom_0_index],
                         bottom_0_diff + ((offset + xw_floor) * width_ + yw_ceil  ));
    caffe_gpu_atomic_add((Dtype) theta_data[ index_x ] *theta_data[ index_y ] *top_diff[bottom_0_index], 
                         bottom_0_diff + ((offset + xw_ceil ) * width_ + yw_ceil  ));
  }
}
 

template <typename Dtype>
void WarpLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data_0 = bottom[0]->gpu_data(); // image
  const Dtype* bottom_data_1 = bottom[1]->gpu_data(); // optical flow
  Dtype* top_data = top[0]->mutable_gpu_data();
  Dtype* theta_data = theta.mutable_gpu_data();
  Dtype* theta_data_ = theta_.mutable_gpu_data();
  Dtype* x_w_data = x_w.mutable_gpu_data();
  const int num_kernels = num_ * channels_ * height_ * width_;
  caffe_gpu_set(bottom[0]->count(), (Dtype)0., top_data);
  switch (outliers_) {
    case WarpParameter_WarpType_TRUNCATE:
      truncate_interp2_fwd<Dtype><<<CAFFE_GET_BLOCKS(num_kernels), CAFFE_CUDA_NUM_THREADS>>>
        (num_kernels, bottom_data_0, bottom_data_1, num_, channels_, height_, width_, 
         theta_data, theta_data_, x_w_data, top_data);
      break;
    case WarpParameter_WarpType_NEAREST:
      nearest_interp2_fwd<Dtype><<<CAFFE_GET_BLOCKS(num_kernels), CAFFE_CUDA_NUM_THREADS>>>
        (num_kernels, bottom_data_0, bottom_data_1, num_, channels_, height_, width_, 
         theta_data, theta_data_, x_w_data, top_data);
      break;

  }
  CUDA_POST_KERNEL_CHECK;
}

template <typename Dtype>
void WarpLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  if (propagate_down[0] || propagate_down[1]) {
    caffe_gpu_set(bottom[0]->count(), (Dtype)0., bottom[0]->mutable_gpu_diff());
    caffe_gpu_set(bottom[1]->count(), (Dtype)0., bottom[1]->mutable_gpu_diff());
    const Dtype* theta_data = theta.mutable_gpu_data();
    const Dtype* theta_data_ = theta_.mutable_gpu_data();
    const Dtype* x_w_data = x_w.mutable_gpu_data();
    const Dtype* top_data = top[0]->gpu_data();
    const Dtype* bottom_0_data = bottom[0]->gpu_data();
    const Dtype* bottom_1_data = bottom[1]->gpu_data();
    const Dtype* top_diff = top[0]->gpu_diff();
    Dtype* bottom_0_diff = bottom[0]->mutable_gpu_diff();
    Dtype* bottom_1_diff = bottom[1]->mutable_gpu_diff();
    const int num_kernels = num_ * channels_ * height_ * width_;
    switch (outliers_) {
      case WarpParameter_WarpType_NEAREST:
        nearest_interp2_bwd<Dtype><<<CAFFE_GET_BLOCKS(num_kernels), CAFFE_CUDA_NUM_THREADS>>>
            (num_kernels, num_, channels_, height_, width_, theta_data, theta_data_, x_w_data, 
             bottom_0_diff, bottom_1_diff, top_diff, top_data, bottom_0_data);
        break;
      case WarpParameter_WarpType_TRUNCATE:
        truncate_interp2_bwd<Dtype><<<CAFFE_GET_BLOCKS(num_kernels), CAFFE_CUDA_NUM_THREADS>>>
            (num_kernels, num_, channels_, height_, width_, theta_data, theta_data_, x_w_data, 
             bottom_0_diff, bottom_1_diff, top_diff, top_data, bottom_0_data);
        break;
    }
    CUDA_POST_KERNEL_CHECK;
    //caffe_gpu_mul(top[0]->count(), top_diff, bottom_0_diff, bottom_0_diff);
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(WarpLayer);

}  // namespace caffe
